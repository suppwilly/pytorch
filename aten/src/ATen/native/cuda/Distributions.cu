#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGenerator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

/**
 * Note [Register spilling in hiprand call for CUDA < 10]
 * ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
 * For CUDA < 10, hiprandStatePhilox4_32_10_t engine achieves poor performance (60% SOL bandwidth)
 * when called to generate one random number at a time. This is because the line
 *            unsigned ret = (&state->output.x)[state->STATE++];
 * in
 *            QUALIFIERS unsigned int hiprand(hiprandStatePhilox4_32_10_t *state)
 * in hiprand/hiprand_kernel.h dynamically indexes into state.output, preventing the compiler from ever
 * storing state.output in registers.
 *
 * CUDA 10 fixed this problem. However, for backwards compatibility, in the following kernels
 * we are using hiprand distributions that utilize curand4 call. curand4 call doesn't have the
 * register spilling problem.
 */

namespace {

template <typename scalar_t>
void poisson_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& lambda,
    std::pair<uint64_t, uint64_t> seeds) {
  at::TensorIterator iter;
  iter.add_output(ret);
  iter.add_input(lambda);
  iter.build();
  at::native::gpu_kernel(iter,
    [seeds] GPU_LAMBDA (scalar_t lambda) -> scalar_t {
      #ifdef __CUDA_ARCH__
      hiprandStatePhilox4_32_10_t state;
      hiprand_init(
          seeds.first,
          blockIdx.x * blockDim.x + threadIdx.x,
          seeds.second,
          &state);
      return static_cast<scalar_t>(hiprand_poisson(&state, lambda));
      #else
      return lambda;  // useless
      #endif
    });
}

struct curand_uniform_wrapper {
  hiprandStatePhilox4_32_10_t &state;
  __device__ curand_uniform_wrapper(hiprandStatePhilox4_32_10_t &state): state(state) {}
  __device__ float operator()() {
    return hiprand_uniform(&state);
  }
};

struct curand_normal_wrapper {
  hiprandStatePhilox4_32_10_t &state;
  __device__ curand_normal_wrapper(hiprandStatePhilox4_32_10_t &state): state(state) {}
  __device__ float operator()() {
    return hiprand_normal(&state);
  }
};

template <typename scalar_t>
void gamma_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& alpha,
    std::pair<uint64_t, uint64_t> seeds) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::TensorIterator iter;
  iter.add_output(ret);
  iter.add_input(alpha);
  iter.build();

  at::native::gpu_kernel(iter,
    [seeds] GPU_LAMBDA (scalar_t alpha) {
      #ifdef __CUDA_ARCH__
      hiprandStatePhilox4_32_10_t state;
      hiprand_init(
          seeds.first,
          blockIdx.x * blockDim.x + threadIdx.x,
          seeds.second,
          &state);

      auto uniform_lambda = curand_uniform_wrapper(state);
      BaseSampler<accscalar_t, decltype(uniform_lambda)> standard_uniform(uniform_lambda);

      auto normal_lambda = curand_normal_wrapper(state);
      BaseSampler<accscalar_t, decltype(normal_lambda)> standard_normal(normal_lambda);
      auto sample = sample_gamma<scalar_t, accscalar_t, decltype(uniform_lambda), decltype(normal_lambda)>(alpha, standard_uniform, standard_normal);
      auto min_value = std::numeric_limits<scalar_t>::min();
      return (min_value > sample) ? min_value : sample;
      #else
      return alpha;  //useless
      #endif
    });
}

template <typename scalar_t>
void gamma_grad_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& self,
    const at::Tensor& output) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(
      ret, self, output,
      [] __device__ (scalar_t& ret_val, const scalar_t& self_val, const scalar_t &output_val) {
        ret_val = standard_gamma_grad_one<scalar_t, accscalar_t>(self_val, output_val);
      });
}

template <typename scalar_t>
void dirichlet_grad_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& x,
    const at::Tensor& alpha,
    const at::Tensor& total) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::cuda::CUDA_tensor_apply4<scalar_t, scalar_t, scalar_t, scalar_t>(
      ret, x, alpha, total,
      [] __device__ (scalar_t& ret_val, const scalar_t& x_val, const scalar_t& alpha_val, const scalar_t& total_val) {
        ret_val = dirichlet_grad_one<scalar_t, accscalar_t>(x_val, alpha_val, total_val);
      });
}

template<typename scalar_t>
void dirichlet_scalar_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& gamma) {
  auto gamma_sum = gamma.sum(-1, true).expand(ret.sizes());
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(ret, gamma, gamma_sum,
  [] __device__(scalar_t &ret_val, const scalar_t &gamma, const scalar_t &gamma_sum) {
    ret_val = gamma / gamma_sum;
    auto min_value = std::numeric_limits<scalar_t>::min();
    auto max_value = 1 - std::numeric_limits<scalar_t>::epsilon();
    ret_val = (min_value > ret_val) ? min_value : ret_val;
    ret_val = (max_value < ret_val) ? max_value : ret_val;
  });
}

} // namespace

namespace at { namespace native {

Tensor _s_poisson_cuda(const Tensor& lambda, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(20);
  }
  Tensor ret = at::empty(lambda.sizes(), lambda.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "poisson_cuda", [&] {
    poisson_cuda_kernel<scalar_t>(ret, lambda, rng_engine_inputs);
  });
  return ret;
}

Tensor _s_gamma_cuda(const Tensor& alpha, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "gamma_cuda", [&] {
     gamma_cuda_kernel<scalar_t>(ret, alpha, rng_engine_inputs);
   });
  return ret;
}

Tensor _s_dirichlet_cuda(const Tensor& alpha, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "dirichlet", [&] {
    Tensor gamma = at::empty(alpha.sizes(), alpha.options());
    gamma_cuda_kernel<scalar_t>(gamma, alpha, rng_engine_inputs);
    dirichlet_scalar_cuda_kernel<scalar_t>(ret, gamma);
  });
  return ret;
}

Tensor _standard_gamma_grad_cuda(const Tensor& self, const Tensor& output) {
  Tensor ret = at::empty(self.sizes(), self.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "_standard_gamma_grad_cuda", [&] {
     gamma_grad_cuda_kernel<scalar_t>(ret, self, output);
   });
  return ret;
}

Tensor _dirichlet_grad_cuda(const Tensor& x, const Tensor& alpha, const Tensor& total) {
  Tensor ret = at::empty(x.sizes(), x.options());
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "_dirichlet_grad_cuda", [&] {
    dirichlet_grad_cuda_kernel<scalar_t>(ret, x, alpha, total);
  });
  return ret;
}

}} // namespace at::native
